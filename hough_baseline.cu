#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////////////////////////////
// ECE 759 Project : Implementation of Hough Transform on Nvidia GPU
//
// The following open source codebases were used to help the development of the project:
//
// https://rosettacode.org/w/index.php?title=Canny_edge_detector&oldid=329226 (last visited December 10, 2022) under CC BY-SA 4.0 License
// https://github.com/nothings/stb.git (last visited December 10, 2022) under MIT License 
// https://github.com/eToTheEcs/hough-transform (last visited December 14, 2022)  -> Code used as a reference model for initial debugging of the algorithm.



#include <iostream>
#include <cmath>
#include <vector>
#include <list>
#include <utility>
#include <queue>
#include <cstring>
#include <hip/hip_runtime.h>
#include <time.h>
#include <chrono>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"
#include "canny.h"



using namespace std;

using std::chrono::high_resolution_clock;
using std::chrono::duration;

typedef unsigned char uchar;

#define PI 3.141592653589793238


#define BIN_WIDTH 1                
#define NUM_BINS 180 / BIN_WIDTH   

//Parameters for Canny Edge detection//

#define KERNEL_SIZE 3

#define RATIO 3



void color_index(uchar* input, int i, int j, int w) {


	input[3*(i*w +j)] = 255;
	input[3*(i*w +j)+1] =0;
	input[3*(i*w +j)+2] = 0;

}	

void draw_line(uchar* input, int w, int h, pair<int, int> p1, pair<int, int> p2) {

int x1, x2, y1, y2;
int thickness = 2;
float gran_x=0.01;
float minx, maxx;
float slope, c;

	x1= p1.first;
	y1= p1.second;
	x2= p2.first;
	y2= p2.second;
	
	slope =   (float)(y2-y1)/(x2-x1);
	c= y1 - slope*x1;




	
	if(x1!=x2) {


		if(x1<x2) {
			minx=(float)x1;
			maxx=(float)x2;
		} else {
			minx=(float)x2;
			maxx=(float)x1;
		}

		float iter;
		int i; //row number 
		int j; //column number

		if(minx<0) {
		
			minx= 0;
		}

		if(maxx>w) {
			maxx=w;
		}
			
		iter= minx;
	

        	
		while(iter < maxx) {

			i= round(c + slope*iter);
			j= round(iter);

			if((i>0)&&(i<h)) {
				color_index(input, i, j, w);
			}

			iter= iter+ gran_x;
			
		}
	} else {
		int maxy, miny;

		if(y2>y1){
			miny=y1;
			maxy=y2;
		} else {
			miny=y2;
			maxy=y1;
		}
	
		if(miny<0) {
			miny= 0;
		}
		if(maxy>h) {
			maxy=h;
		}

		for(int i=miny; i<maxy; i++) {
			color_index(input, i, x1, w);
		}


	}
}




void canny_detector(const uchar* input, const int w, const int h, const  int threshold, uchar* output) {

	bitmap_info_header_t bmp_ih;
	pixel_t* input_int;
	pixel_t* output_int;

	
	input_int= new pixel_t [w*h];
	output_int = new pixel_t [w*h];

	for(int i=0; i< w; i++) {
		for(int j=0; j<h; j++) {
			input_int[j*w + i] = input[j*w + i];
		}
	}
	
	bmp_ih.width = w;
        bmp_ih.height = h;
	bmp_ih.bmp_bytesz = w*h;

	output_int = canny_edge_detection(input_int, &bmp_ih, threshold,  threshold*RATIO, 1); 

	for(int i=0; i< w; i++) {
		for(int j=0; j<h; j++) {
			output[j*w + i] = (uchar)output_int[j*w + i];
		}
	}

	
}




void polarToCartesian(int rho, int theta, pair<int,int> &p1, pair<int,int> &p2);

__global__ void voting_kernel(uchar * edge_array, int* hough_space,  int width, int height, int diag) {

	//Pixel indices
	int i, j;
	
	int rho;
	double theta_rad;
	

	int index = blockDim.x*blockIdx.x + threadIdx.x;

	i = index/width;
	j = index%width;

	if(index< width* height){
		if(*(edge_array + index) == 255) {
		       for(int theta = 0; theta <= 180; theta += BIN_WIDTH) {

				theta_rad = ((double)((double)(theta -90) /180)) * PI;
                       		rho = round(j * cos((double)(theta_rad)) + i * sin((double)(theta_rad))) + diag;

				//Atomically vote in the hough space
                       		atomicAdd((hough_space + rho*(NUM_BINS+1) + theta), 1);
			}
                }
			
	}
}
	

__global__ void equation_forming_kernel (int* hough_space, int input_length, int* rho_vals, int* theta_vals, int* output_length, int threshold, int diag) {

	
	extern __shared__ int s [];

	//Shared Memory Arrays
        int* arr_size = &s[0];

        int* rho_arr =  &s[1];   
        int* theta_arr =  &s[1+ blockDim.x];



        int rho, theta;
	int smem_array_size;
	
	

	int store_index;	

	int index = blockDim.x*blockIdx.x + threadIdx.x;
	
	if(threadIdx.x ==0) {
		//Shared Memory variables are always uninitialized, initialize it here:
 		*arr_size=0;
		
	}

	__syncthreads();

	if(index< input_length) {
		if(*(hough_space + index) > threshold) {
			rho =   index/(NUM_BINS+1);
			theta = index%(NUM_BINS+1);
			
			rho = rho - diag;
			theta = theta -90;
			
			store_index = atomicAdd(arr_size, 1);
				
			rho_arr[store_index] = rho ;
			theta_arr[store_index] = theta;
		 }
	}
			 
			
	__syncthreads();
	
	smem_array_size = *arr_size;
	
	if(threadIdx.x ==0) {
		*arr_size = atomicAdd(output_length, smem_array_size);
	}
        __syncthreads();
     
	if(threadIdx.x < smem_array_size){
		rho_vals[(*arr_size)+ threadIdx.x] = rho_arr[threadIdx.x];
		theta_vals[(*arr_size)+ threadIdx.x] = theta_arr[threadIdx.x];
	}
} 

	      	

	

	

void reference_serial_voting( const uchar* edges, int w, int h, int * hough_space)  {

 
    int rho;
    int theta;
    int diag;

    diag = hypot(h, w);
    double theta_rad;
 
    

    

    

    for(int i = 0; i < h; ++i) {
        for(int j = 0; j < w; ++j) {
	   
            if(edges[i*w+j] == 255) {  

		//In the Image:
                // We are taking theta from -90 to +90. Since negative indices are not allowed theta is offsetted to go from 0 to 180
		// The value of rho can go from (-max(WIDTH, HEIGHT), sqrt(WIDTH^2 + HEIGHT^2)) -> Taking (-sqrt(WIDTH^2, HEIGHT^2), sqrt(WIDTH^2, HEIGHT^2)) to be on safer side. -> Since negative indices are not alllowed rho here can go from 0 to 2*(sqrt(WIDTH^2 + HEIGHT^2)
                for(theta = 0; theta <= NUM_BINS; theta += BIN_WIDTH) {
		    theta_rad = ((double)(theta-90)/180) * PI;
                    rho = round(j * cos(theta_rad)  + i * sin(theta_rad)) + diag;
                    (*(hough_space + rho*(NUM_BINS +1) + theta))++;
                }
            }
        }
    }




}

void reference_equation_forming( int* hough_space ,   list<pair<int,int>> &equations,   int hough_size, int line_threshold) {

    pair<int,int> temp_pair;
    int rho, theta; 		
    int diag = hough_size / (2*(NUM_BINS+1));

   
    for(int i = 0; i < hough_size; ++i) {
       

            if((*(hough_space + i)) > line_threshold) {

		rho= i/(NUM_BINS+1);
 		theta = i%(NUM_BINS+1);
		
		//In the final equations theta -> (-90 to 90); rho -> (-maxdistance, maxdistance)
                rho = rho - diag;
                theta = theta - 90;
		temp_pair = pair<int, int>( rho, theta);
                equations.insert(equations.end(), temp_pair);
                
            }
     }
    
}

void initialize_to_zero ( int** arr, unsigned int size) {
	for( int i=0; i< size; i++) {
		*(*arr + i) =0;
	}
}


int main(int argc, char** argv) {

   
    int theta;      
    int canny_threshold;
    int rho;       
    int diag;
    int rho_max;

    

    hipEvent_t start_cuda;
    hipEvent_t stop_cuda;
    float ms;

    high_resolution_clock::time_point start;
    high_resolution_clock::time_point end;
    duration<double, std::milli> duration_sec;
    
    hipEventCreate(&start_cuda);
    hipEventCreate(&stop_cuda);


    if(argc < 4) {
        printf("USAGE: hough [fileName] [treshold] [canny_threshold]\n");
        return EXIT_FAILURE;
    }

    int line_threshold = atoi(argv[2]);
    canny_threshold = atoi(argv[3]);

    
    ////////////////////////////////////////////////////////////
    ///////////////    Loading Input Image     /////////////////
    ////////////////////////////////////////////////////////////

    int h, w, inp_ch, desired_ch;

    uchar* image_src;
   
    desired_ch =1;

    
    image_src = stbi_load(argv[1], &w, &h, &inp_ch, desired_ch);


    cout<<"Image Loaded:: name= "<<argv[1]<<" w= "<<w<<" h= "<<h<<" inp_ch= "<< inp_ch<<endl; 

    
    (void)stbi_write_jpg("source.jpg",  w, h, 1, image_src, 100);

    //Calculating parameters from the dimensions of the image
    diag = hypot(h, w);
    rho_max = 2*diag;

    /////////////////////////////////////////////////////////////
    //////////////   Canny Edge Detection Stage  ////////////////
    /////////////////////////////////////////////////////////////

    uchar* edges;
    edges = new uchar [w*h];
    
    canny_detector(image_src, w, h, canny_threshold, edges);
        

    (void)stbi_write_jpg("edges.jpg",  w, h, 1, edges, 100);





    ///////////////////////////////////////////////////////////
    //////////// Reference Hough Space Computation/////////////
    ///////////////////////////////////////////////////////////

    int* hough_space_ref;   

    list<pair<int, int>> equations_ref;
    hough_space_ref = new int [ (NUM_BINS+1)* rho_max ]; 
   
    //Important:: Initialize accumulator matrix to 0
    initialize_to_zero(&hough_space_ref, (NUM_BINS+1)* rho_max); 


    start = high_resolution_clock::now();
    //Reference Serial implementation 
    reference_serial_voting( edges, w, h,  hough_space_ref);
    reference_equation_forming( hough_space_ref ,  equations_ref,  rho_max * (NUM_BINS+1), line_threshold);
    end = high_resolution_clock::now();
    

    duration_sec = std::chrono::duration_cast<duration<double, std::milli>>(end-start);
 
    cout<<fixed;
    cout<<"Time taken by serial implementation  "<<duration_sec.count()<<endl;
    
    /////////////////////////////////////////////////////
    /////////////////////////////////////////////////////
    /////////////////////////////////////////////////////


    /////////////////////////////////////////////////////
    ////////     Code For GPU Execution     /////////////
    /////////////////////////////////////////////////////

    uchar* edges_host;
    uchar* edges_device;
    int* hough_space_device;
    int* rho_arr_device;
    int* theta_arr_device;
    int* rho_arr_host;
    int* theta_arr_host;
    int* eq_size_device;
    int* eq_size_host;
    int* hough_space_host;
    
    hipError_t error_code;

    int  number_of_threads = 1024;
    int number_of_blocks_1 = (h * w + number_of_threads -1) /number_of_threads;
    int number_of_blocks_2 = (rho_max*(NUM_BINS+1) + number_of_threads-1) / number_of_threads;


    
    //Allocating GPU side memories
    hipMalloc((void**)&edges_device, sizeof(uchar)*w*h);
    hipMalloc((void**)&hough_space_device, sizeof(int)* rho_max * (NUM_BINS+1));
    hipMalloc((void**)&theta_arr_device, sizeof(int)* rho_max * (NUM_BINS+1));
    hipMalloc((void**)&rho_arr_device, sizeof(int) *rho_max *(NUM_BINS+1));
    hipMalloc((void**)&eq_size_device, sizeof(int));


    //Allocating GPU side memories
    edges_host= new uchar [h*w];
    hough_space_host = new int [rho_max* (NUM_BINS+1)];
    rho_arr_host = new int [rho_max* (NUM_BINS+1)];
    theta_arr_host = new int [rho_max* (NUM_BINS+1)];
    eq_size_host = new int [rho_max *(NUM_BINS+1)];
		

    //Initializing host side global variables.
    hipMemset(hough_space_device, 0, sizeof(int)*rho_max*(NUM_BINS+1));
    hipMemset(eq_size_device, 0, sizeof(int));

    
    
    //Copy the Matrix to the edges_host to pass onto the GPU
    for( int i=0; i< h; i++) {
	for( int j=0; j< w; j++) {
		edges_host[i*w + j] = edges[i*w + j] ;
	}
    }

   

    error_code = hipMemcpy((void*)edges_device, (void*)edges_host, sizeof(uchar)*h*w, hipMemcpyHostToDevice);

    if(error_code !=0){
	cout<<"Copy of Edges Array from host to device Failed, Error code "<<error_code<<endl;
    }
	

    //Launching and Benchmarking the kernels.

    hipEventRecord(start_cuda);
   
    voting_kernel <<<number_of_blocks_1, number_of_threads>>> ( edges_device, hough_space_device, w, h, diag);

    equation_forming_kernel<<<number_of_blocks_2, number_of_threads, sizeof(int)*(2*number_of_threads +1)>>> (hough_space_device, rho_max* (NUM_BINS+1), rho_arr_device, theta_arr_device, eq_size_device, line_threshold, diag) ;

    hipEventRecord(stop_cuda);

    error_code = hipEventSynchronize(stop_cuda);
 
    if(error_code !=0){
	cout<<"Kernel Launch Failed, Error code "<<error_code<<endl;
    }



    hipEventElapsedTime(&ms, start_cuda, stop_cuda);


    //Copying the results back
    error_code = hipMemcpy((void*)eq_size_host, (void*)eq_size_device, sizeof(int)*1, hipMemcpyDeviceToHost);

    if(error_code !=0){
	cout<<"Copy of eq_size from device to host Failed, Error code "<<error_code<<endl;
    }


    error_code = hipMemcpy((void*)rho_arr_host, (void*)rho_arr_device, sizeof(int)*rho_max*(NUM_BINS+1), hipMemcpyDeviceToHost);

    if(error_code !=0){
	cout<<"Copy of rho_arr from device to host Failed, Error code "<<error_code<<endl;
    }


    error_code = hipMemcpy((void*)theta_arr_host, (void*)theta_arr_device, sizeof(int)*rho_max*(NUM_BINS+1), hipMemcpyDeviceToHost);

    if(error_code !=0){
	cout<<"Copy of theta_arr from device to host Failed, Error code "<<error_code<<endl;
    }

   hipMemcpy((void*) hough_space_host, (void*)hough_space_device, sizeof(int)*rho_max*(NUM_BINS+1), hipMemcpyDeviceToHost);
       
  
    //Freeing the memories
    hipFree(edges_device);
    hipFree(hough_space_device);
    hipFree(rho_arr_device);
    hipFree(theta_arr_device);

    cout<<"Time taken by GPU implementation  "<<ms<<endl;


    ////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////


 	

    ////////////////////////////////////////////////////////////
    ///////// Plotting Found Equations to the Image ////////////
    ////////////////////////////////////////////////////////////
      
    uchar* o_multi;


    //Making the monochrome image multi-color
    o_multi = new uchar [3*w*h];


    for(int i=0; i<h; i++) {
	for( int j=0; j<w; j++) {
		o_multi[3*(i*w+j)] = image_src[i*w + j];
		o_multi[3*(i*w+j)+1] = image_src[i*w + j];
		o_multi[3*(i*w+j)+2] = image_src[i*w + j];

	}
    }

    

    // Plotting lines from equations   
    //  For Host
    //  list<pair<int,int>>::iterator  it;
    //  for(int i = 0; i < equations_ref.size(); ++i) {
    //
    //   it = equations_ref.begin();

    //   advance(it, i);
    //   rho =  (*it).first;
    //   theta = (*it).second ;

    //   pair<int, int> p1, p2;  
    //   polarToCartesian(rho, theta, p1, p2);

    //   //cout<< p1 << ", " << p2 <<"\n";

    //   
    //    draw_line(o_multi, w,h, p1, p2);

    //}

    // For GPU
    
    //cout<<"Before Line Drawing"<<endl;
    for(int i=0; i< *eq_size_host; i++) {
        rho =  rho_arr_host[i];
        theta = theta_arr_host[i];
        pair<int, int> p1, p2;
        polarToCartesian(rho, theta, p1, p2);
        draw_line(o_multi,w,h, p1, p2);	
    }
  
      

    (void)stbi_write_jpg("output.jpg",  w, h, 3, o_multi, 100);

    	

    //////////////////////////////////////////////////////////
    //////////////////////////////////////////////////////////
    //////////////////////////////////////////////////////////

    ////// Freeing up the memories/////////

    delete[] rho_arr_host;
    delete[] theta_arr_host;
    delete eq_size_host;
    delete[] edges_host;


    return 0;
}


void polarToCartesian(int rho, int theta, pair<int, int> &p1, pair<int, int> &p2) {


	double theta_rad = (((double)theta)/180) * PI;
	int x0 = round(rho * cos(theta_rad));
	int y0 = round(rho * sin(theta_rad));

	p1.first = round(x0 + 5000 * (-1*sin(theta_rad)));
	p1.second = round(y0 + 5000 * (cos(theta_rad)));

	p2.first = round(x0 - 5000 * (-1* sin(theta_rad)));
	p2.second = round(y0 - 5000 * (cos(theta_rad)));
}
